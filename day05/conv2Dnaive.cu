
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;

__global__ void conv2D(float *input, float *kernel, int k, int n, float *output)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    float value = 0;
    for (int i = -n; i <= n; i++)
    {
        for (int j = -n; j <= n; j++)
        {
            if (0 <= x + i && x + i < k && 0 <= y + j && y + j < k)
                value += input[k * (y + j) + x + i] * kernel[(2 * n + 1) * (n + j) + n + i];
        }
    }
    output[k * y + x] = value;
}

int main(void)
{
    // kernel declaration
    int n = 2;
    int kernel_dim = 2 * n + 1;
    int kernel_size = kernel_dim * kernel_dim * sizeof(float);
    float h_kernel[kernel_dim][kernel_dim];
    for (int i = 0; i < kernel_dim; i++)
    {
        for (int j = 0; j < kernel_dim; j++)
        {
            h_kernel[i][j] = 5 - (abs(2 - i) + abs(2 - j));
        }
    }

    // input declaration
    int k = 7;
    int input_size = k * k * sizeof(float);
    float h_input[k][k];
    for (int r = 0; r < k; r++)
    {
        int temp = r + 1;
        for (int c = 0; c < k; c++)
        {
            h_input[r][c] = temp + c;
        }
    }

    cout << "kernel" << endl;
    for (int i = 0; i < kernel_dim; i++)
    {
        for (int j = 0; j < kernel_dim; j++)
        {
            cout << h_kernel[i][j] << " ";
        }
        cout << endl;
    }
    cout << "input" << endl;
    for (int r = 0; r < k; r++)
    {
        for (int c = 0; c < k; c++)
        {
            cout << h_input[r][c] << " ";
        }
        cout << endl;
    }

    float *d_input, *d_kernel, *d_output;
    hipMalloc((void **)&d_input, input_size);
    hipMalloc((void **)&d_kernel, kernel_size);
    hipMalloc((void **)&d_output, input_size);
    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice);

    dim3 block_size(k, k);
    conv2D<<<1, block_size>>>(d_input, d_kernel, k, n, d_output);
    // Synchronize and check for errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        cout << "CUDA Error: " << hipGetErrorString(err) << endl;
        return -1;
    }

    float h_output[input_size];
    hipMemcpy(h_output, d_output, input_size, hipMemcpyDeviceToHost);

    cout << "Output" << endl;
    for (int i = 0; i < k; i++)
    {
        for (int j = 0; j < k; j++)
        {
            cout << h_output[k * i + j] << " ";
        }
        cout << endl;
    }

    // Free the memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);

    return 0;
}