#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA-capable devices found.\n");
        return 0;
    }

    for (int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, dev); 

        printf("Device %d: %s\n", dev, prop.name);
        if (prop.managedMemory)
        {
            printf("  ✅ Supports Unified Memory\n");
        }
        else
        {
            printf("  ❌ Does NOT support Unified Memory\n");
        }
    }

    return 0;
}